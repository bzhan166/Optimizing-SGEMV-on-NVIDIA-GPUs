#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include "utils.h"
#include <unistd.h>

int main(int argc, char *argv[])
{
    if (argc != 4) {
        printf("please input [m] [n] [kernel_num].\n");
        printf("kernel_num == 1: mysgemv (default).\n");
        printf("kernel_num == 2: cuBLAS SGEMV.\n");
        exit(-1);
    }
    int m, n, kernel_num = 1;
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    kernel_num = atoi(argv[3]);
    if (m != (m & -32) || n != (n & -32)) {
        printf("currently we only support m, n divisible by 32.\n");
        printf("rounded m, n to multipliers of 32.\n");
        m = (m & -32); n = (n & -32);
    }
    if ( (kernel_num!=1&&kernel_num!=2) || m <= 0 || n <= 0 ) {
        printf("Illegal input, returned.\n");
        exit(-1);
    }
    printf("m = %d, n = %d.\n", m, n);
    if (kernel_num == 1) printf("Testing my sgemv.\n");
    else printf("Testing cuBLAS SGEMV.\n");
    float *hA, *hX, *hY, *hY_ref;
    float *dA, *dX, *dY, *dY_ref;
    float elapsed_time;
    hA = (float*)malloc(sizeof(float) * m * n);
    hX = (float*)malloc(sizeof(float) * n);
    hY = (float*)malloc(sizeof(float) * m);
    hY_ref = (float*)malloc(sizeof(float) * m);
    float alpha = 1., beta = 1.;
    int N = 5;
    randomize_matrix(hA, m, n);
    randomize_matrix(hX, n, 1);
    randomize_matrix(hY, m, 1);
    randomize_matrix(hY_ref, m, 1);
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);
    CUDA_CALLER(hipMalloc ((void**)&dA, sizeof(float) * m * n));
    CUDA_CALLER(hipMalloc ((void**)&dX, sizeof(float) * n));
    CUDA_CALLER(hipMalloc ((void**)&dY, sizeof(float) * m));
    CUDA_CALLER(hipMalloc ((void**)&dY_ref, sizeof(float) * m));
    CUDA_CALLER(hipMemcpy(dA, hA, sizeof(float) * m * n, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dX, hX, sizeof(float) * n, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dY, hY, sizeof(float) * m, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dY_ref, hY_ref, sizeof(float) * m, hipMemcpyHostToDevice));
    hipblasHandle_t myHandle; hipblasCreate(&myHandle);

    if (kernel_num == 1){
        printf("Start the sanity check...\n");
        fflush(stdout);
        mysgemv(m, n, alpha, dA, m, dX, beta, dY);
        hipblasSgemv(myHandle, HIPBLAS_OP_N, m, n, &alpha, dA, m, dX, 1, &beta, dY_ref, 1);
        
        hipDeviceSynchronize();
        hipMemcpy(hY, dY, sizeof(float)*m, hipMemcpyDeviceToHost);
        hipMemcpy(hY_ref, dY_ref, sizeof(float)*m, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        if (!verify_matrix(hY, hY_ref, m)){
            printf("did not pass the sanity check, returned.\n");
            exit(-2);
        }else{
            printf("Sanity check passed. Start performance benchmarking...\n");
            fflush(stdout);
        }
    }

    hipDeviceSynchronize();
    hipEventRecord(beg);
    if (kernel_num == 1){
        for (int i = 0; i < N; i++){
            mysgemv(m, n, alpha, dA, m, dX, beta, dY);
        }
    }else{
        for (int i = 0; i < N; i++){
            hipblasSgemv(myHandle, HIPBLAS_OP_N, m, n, &alpha, dA, m, dX, 1, &beta, dY, 1);
        }
    }

    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1000.;
    printf("Average elasped time: %f second, performance: %f GFLOPS.\n", elapsed_time/N,2.*N*1e-9*m*n/elapsed_time);
    hipDeviceSynchronize();
    free(hA);free(hX);free(hY);free(hY_ref);
    hipFree(dA);hipFree(dX);hipFree(dY);hipFree(dY_ref);
    hipDeviceSynchronize();
    return 0;
}